#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------------------------------
// Copyrighted by Marko Rakita.
// Author: Marko Rakita
// File contains: Mocked neural network softmax layer, used in tests.
// Created: 02/20/2016.
// ----------------------------------------------------------------------------------------------------

#include "include/mocksoftmaxlayer.cuh"

#include <hip/hip_runtime.h>

#include "../../../../neuralnetwork/layers/include/outputlayer.cuh"
#include "../../../../utils/include/asserts.cuh"
#include "../../../../utils/include/cudaasserts.cuh"

MockSoftMaxLayer::MockSoftMaxLayer(uint inputDataSize, uint inputDataCount)
	:
	SoftMaxLayer(ParallelismMode::Model, 0, 0, inputDataSize, inputDataCount, true)
{
}

void MockSoftMaxLayer::AllocateBuffers(bool allocateTrainBuffers)
{
	// Allocating input data buffer.
	if (m_holdsInputData)
	{
		CudaAssert(hipHostMalloc<float>(&m_inputDataBuffer, m_inputBufferSize));
	}

	// Allocating input activations maximums buffer.
	CudaAssert(hipHostMalloc<float>(&m_inputActivationsMaxBuffer, m_inputDataCount * sizeof(float)));

	// Allocating input activations maximums buffer.
	CudaAssert(hipHostMalloc<float>(&m_exponentialsSumBuffer, m_inputDataCount * sizeof(float)));

	// Allocating activation data buffers.
	CudaAssert(hipHostMalloc<float>(&m_activationDataBuffer, m_activationBufferSize));

	// Allocating buffers necessary for training.
	if (allocateTrainBuffers)
	{
		// Allocating input gradients buffer.
		CudaAssert(hipHostMalloc<float>(&m_inputGradientsBuffer, m_inputBufferSize));
	}
}

MockSoftMaxLayer::~MockSoftMaxLayer()
{
	if (m_holdsInputData && m_inputDataBuffer != NULL)
	{
		CudaAssert(hipHostFree(m_inputDataBuffer));
		m_inputDataBuffer = NULL;
	}
	if (m_inputGradientsBuffer != NULL)
	{
		CudaAssert(hipHostFree(m_inputGradientsBuffer));
		m_inputGradientsBuffer = NULL;
	}
	if (m_inputActivationsMaxBuffer != NULL)
	{
		CudaAssert(hipHostFree(m_inputActivationsMaxBuffer));
		m_inputActivationsMaxBuffer = NULL;
	}
	if (m_exponentialsSumBuffer != NULL)
	{
		CudaAssert(hipHostFree(m_exponentialsSumBuffer));
		m_exponentialsSumBuffer = NULL;
	}
	if (m_activationDataBuffer != NULL)
	{
		CudaAssert(hipHostFree(m_activationDataBuffer));
		m_activationDataBuffer = NULL;
	}
	if (m_holdsActivationGradients && m_activationGradientsBuffer != NULL)
	{
		CudaAssert(hipHostFree(m_activationGradientsBuffer));
		m_activationGradientsBuffer = NULL;
	}
	if (m_NLLsBuffer != NULL)
	{
		CudaAssert(hipHostFree(m_NLLsBuffer));
		m_NLLsBuffer = NULL;
	}
}

void MockSoftMaxLayer::LoadInputs()
{
	ShipAssert(m_prevLayers.size() == 1, "We do not support more than one previous layer in tests, for now.");
	CudaAssert(hipMemcpy(m_inputDataBuffer, m_prevLayers[0]->GetActivationDataBuffer(), m_inputBufferSize, hipMemcpyDeviceToHost));
}

void MockSoftMaxLayer::LoadActivationGradients()
{
	ShipAssert(m_nextLayers.size() == 1, "We do not support more than one previous layer in tests, for now.");

	CudaAssert(hipHostMalloc<float>(&m_activationGradientsBuffer, m_activationBufferSize));
	CudaAssert(hipMemcpy(m_activationGradientsBuffer, m_nextLayers[0]->GetInputGradientsBuffer(), m_activationBufferSize, hipMemcpyDeviceToHost));
}

void MockSoftMaxLayer::StabilizeInputs()
{
	// Finding maximums of input activations.
	for (uint dataIndex = 0; dataIndex < m_inputDataCount; ++dataIndex)
	{
		m_inputActivationsMaxBuffer[dataIndex] = m_inputDataBuffer[dataIndex];
		for (uint activationIndex = 1; activationIndex < m_activationDataSize; ++activationIndex)
		{
			m_inputActivationsMaxBuffer[dataIndex] = max(m_inputActivationsMaxBuffer[dataIndex], m_inputDataBuffer[activationIndex * m_inputDataCount + dataIndex]);
		}
	}

	// Substracting maximums of input activations from all the input activations.
	for (uint dataIndex = 0; dataIndex < m_inputDataCount; ++dataIndex)
	{
		for (uint activationIndex = 0; activationIndex < m_activationDataSize; ++activationIndex)
		{
			m_activationDataBuffer[activationIndex * m_inputDataCount + dataIndex] = m_inputDataBuffer[activationIndex * m_inputDataCount + dataIndex] -
				m_inputActivationsMaxBuffer[dataIndex];
		}
	}
}

void MockSoftMaxLayer::CalculateSoftMaximums()
{
	// Computing the exponentials.
	for (uint dataIndex = 0; dataIndex < m_inputDataCount; ++dataIndex)
	{
		for (uint activationIndex = 0; activationIndex < m_activationDataSize; ++activationIndex)
		{
			m_activationDataBuffer[activationIndex * m_inputDataCount + dataIndex] = exp(m_activationDataBuffer[activationIndex * m_inputDataCount + dataIndex]);
		}
	}

	// Computing sum of the exponentials.
	for (uint dataIndex = 0; dataIndex < m_inputDataCount; ++dataIndex)
	{
		m_exponentialsSumBuffer[dataIndex] = m_activationDataBuffer[dataIndex];
		for (uint activationIndex = 1; activationIndex < m_activationDataSize; ++activationIndex)
		{
			m_exponentialsSumBuffer[dataIndex] += m_activationDataBuffer[activationIndex * m_inputDataCount + dataIndex];
		}
	}

	// Dividing exponentials with their sum to get soft maximums.
	for (uint dataIndex = 0; dataIndex < m_inputDataCount; ++dataIndex)
	{
		for (uint activationIndex = 0; activationIndex < m_activationDataSize; ++activationIndex)
		{
			m_activationDataBuffer[activationIndex * m_inputDataCount + dataIndex] /= m_exponentialsSumBuffer[dataIndex];
		}
	}
}

void MockSoftMaxLayer::CalculateNegativeLogLikelihoods(uint* dataLabels)
{
	if (m_NLLsBuffer == NULL)
	{
		CudaAssert(hipHostMalloc<float>(&m_NLLsBuffer, m_inputDataCount * sizeof(float)));
	}

	for (uint dataIndex = 0; dataIndex < m_inputDataCount; ++dataIndex)
	{
		m_NLLsBuffer[dataIndex] = m_inputActivationsMaxBuffer[dataIndex] + logf(m_exponentialsSumBuffer[dataIndex]) -
			m_inputDataBuffer[dataLabels[dataIndex] * m_inputDataCount + dataIndex];
	}
}

void MockSoftMaxLayer::DoForwardProp(PropagationMode propagationMode)
{
	StabilizeInputs();
	CalculateSoftMaximums();

	if (m_nextLayers[0]->GetLayerType() == LayerType::Output)
	{
		OutputLayer* outputLayer = static_cast<OutputLayer*>(m_nextLayers[0]);
		if (outputLayer->GetLossFunctionType() == LossFunctionType::CrossEntropy)
		{
			uint* tempHostLabelsBuffer;
			size_t labelsBufferSize = m_inputDataCount * sizeof(uint);
			CudaAssert(hipHostMalloc<uint>(&tempHostLabelsBuffer, labelsBufferSize));
			CudaAssert(hipMemcpy(tempHostLabelsBuffer, outputLayer->GetDataLabels(), labelsBufferSize, hipMemcpyDeviceToHost));

			CalculateNegativeLogLikelihoods(tempHostLabelsBuffer);

			CudaAssert(hipHostFree(tempHostLabelsBuffer));
		}
		else
		{
			ShipAssert(false, "Currently not supported!");
		}
	}
}

void MockSoftMaxLayer::CrossEntropyBackwardProp(uint* dataLabels)
{
	for (uint activationIndex = 0; activationIndex < m_activationDataSize; ++activationIndex)
	{
		for (uint dataIndex = 0; dataIndex < m_inputDataCount; ++dataIndex)
		{
			uint activationsOffset = activationIndex * m_inputDataCount + dataIndex;
			m_inputGradientsBuffer[activationsOffset] = m_activationDataBuffer[activationsOffset] - (dataLabels[dataIndex] == activationIndex ? 1.f : 0.f);
		}
	}
}

void MockSoftMaxLayer::DoBackwardProp()
{
	if (m_nextLayers[0]->GetLayerType() == LayerType::Output)
	{
		OutputLayer* outputLayer = static_cast<OutputLayer*>(m_nextLayers[0]);
		if (outputLayer->GetLossFunctionType() == LossFunctionType::CrossEntropy)
		{
			uint* tempHostLabelsBuffer;
			size_t labelsBufferSize = m_inputDataCount * sizeof(uint);
			CudaAssert(hipHostMalloc<uint>(&tempHostLabelsBuffer, labelsBufferSize));
			CudaAssert(hipMemcpy(tempHostLabelsBuffer, outputLayer->GetDataLabels(), labelsBufferSize, hipMemcpyDeviceToHost));

			CrossEntropyBackwardProp(tempHostLabelsBuffer);

			CudaAssert(hipHostFree(tempHostLabelsBuffer));
		}
		else
		{
			ShipAssert(false, "Currently not supported!");
		}
	}
	else
	{
		ShipAssert(false, "Currently not supported!");
	}
}