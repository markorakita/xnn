// ----------------------------------------------------------------------------------------------------
// Copyrighted by Marko Rakita.
// Author: Marko Rakita
// File contains: Mocked neural network dropout layer, used in tests.
// Created: 02/16/2016.
// ----------------------------------------------------------------------------------------------------

#include "include/mockdropoutlayer.cuh"

#include <chrono>

#include <hip/hip_runtime.h>

#include "../../../../utils/include/asserts.cuh"
#include "../../../../utils/include/cudaasserts.cuh"

MockDropoutLayer::MockDropoutLayer(uint inputNumChannels, uint inputDataWidth, uint inputDataHeight, uint inputDataCount, float dropProbability,
	hiprandState* curandStatesBuffer)
	:
	DropoutLayer(ParallelismMode::Model, 0, 0, curandStatesBuffer, 0, 1, inputNumChannels, inputDataWidth, inputDataHeight, inputDataCount, true,
		dropProbability, false, true)
{
}

void MockDropoutLayer::AllocateBuffers(bool allocateTrainBuffers)
{
	// Allocating input data buffer.
	if (m_holdsInputData)
	{
		CudaAssert(hipHostMalloc<float>(&m_inputDataBuffer, m_inputBufferSize));
	}

	// Allocating dropout filter buffer.
	CudaAssert(hipHostMalloc<float>(&m_dropoutFilter, m_dropoutFilterSize));

	// Allocating activation data buffers.
	CudaAssert(hipHostMalloc<float>(&m_activationDataBuffer, m_activationBufferSize));

	// Allocating buffers necessary for training.
	if (allocateTrainBuffers)
	{
		// Allocating input gradients buffer.
		CudaAssert(hipHostMalloc<float>(&m_inputGradientsBuffer, m_inputBufferSize));

		// Allocating activation gradients buffer.
		if (m_holdsActivationGradients)
		{
			CudaAssert(hipHostMalloc<float>(&m_activationGradientsBuffer, m_activationBufferSize));
		}
	}
}

MockDropoutLayer::~MockDropoutLayer()
{
	if (m_holdsInputData && m_inputDataBuffer != NULL)
	{
		CudaAssert(hipHostFree(m_inputDataBuffer));
		m_inputDataBuffer = NULL;
	}
	if (m_inputGradientsBuffer != NULL)
	{
		CudaAssert(hipHostFree(m_inputGradientsBuffer));
		m_inputGradientsBuffer = NULL;
	}
	if (m_dropoutFilter != NULL)
	{
		CudaAssert(hipHostFree(m_dropoutFilter));
		m_dropoutFilter = NULL;
	}
	if (m_activationDataBuffer != NULL)
	{
		CudaAssert(hipHostFree(m_activationDataBuffer));
		m_activationDataBuffer = NULL;
	}
	if (m_holdsActivationGradients && m_activationGradientsBuffer != NULL)
	{
		CudaAssert(hipHostFree(m_activationGradientsBuffer));
		m_activationGradientsBuffer = NULL;
	}
}

void MockDropoutLayer::LoadInputs()
{
	ShipAssert(m_prevLayers.size() == 1, "We do not support more than one previous layer in tests, for now.");
	CudaAssert(hipMemcpy(m_inputDataBuffer, m_prevLayers[0]->GetActivationDataBuffer(), m_inputBufferSize, hipMemcpyDeviceToHost));
}

void MockDropoutLayer::LoadActivationGradients()
{
	ShipAssert(m_nextLayers.size() == 1, "We do not support more than one previous layer in tests, for now.");

	CudaAssert(hipMemcpy(m_activationGradientsBuffer, m_nextLayers[0]->GetInputGradientsBuffer(), m_activationBufferSize, hipMemcpyDeviceToHost));
}

void MockDropoutLayer::CreateDropoutFilter()
{
	// Filling dropout filter with random values.
	float* deviceBuffer;
	CudaAssert(hipMalloc<float>(&deviceBuffer, m_dropoutFilterSize));

	uint dropoutFilterLength = (uint)(m_dropoutFilterSize / sizeof(float));
	InitializeBufferFromUniformDistribution(deviceBuffer, dropoutFilterLength, 0.f, 1.f, m_curandStatesBuffer);
	SynchronizeCalculations();

	CudaAssert(hipMemcpy(m_dropoutFilter, deviceBuffer, m_dropoutFilterSize, hipMemcpyDeviceToHost));
	CudaAssert(hipFree(deviceBuffer));

	// Dropping filter values which are not above the drop probability.
	for (uint i = 0; i < dropoutFilterLength; ++i)
	{
		m_dropoutFilter[i] = m_dropoutFilter[i] > m_dropProbability ? 1.0f : 0.0f;
	}
}

void MockDropoutLayer::ApplyDropoutFilter()
{
	size_t dropoutFilterLength = m_dropoutFilterSize / sizeof(float);
	for (size_t i = 0; i < dropoutFilterLength; ++i)
	{
		m_activationDataBuffer[i] = m_inputDataBuffer[i] * m_dropoutFilter[i];
	}
}

void MockDropoutLayer::DoForwardProp(PropagationMode propagationMode)
{
	CreateDropoutFilter();
	ApplyDropoutFilter();
}

void MockDropoutLayer::DoBackwardProp()
{
	size_t dropoutFilterLength = m_dropoutFilterSize / sizeof(float);
	for (size_t i = 0; i < dropoutFilterLength; ++i)
	{
		m_inputGradientsBuffer[i] = m_activationGradientsBuffer[i] * m_dropoutFilter[i];
	}
}