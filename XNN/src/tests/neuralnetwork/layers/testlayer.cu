#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------------------------------
// Copyrighted by Marko Rakita.
// Author: Marko Rakita
// File contains: Tests for abstract layer.
// Created: 12/07/2020.
// ----------------------------------------------------------------------------------------------------

#include "include/testlayer.cuh"

#include <hip/hip_runtime.h>

#include "mock/include/mockinputlayer.cuh"
#include "../../../neuralnetwork/include/neuralnet.cuh"
#include "../../../utils/include/asserts.cuh"
#include "../../../utils/include/cudaasserts.cuh"

TestLayer::TestLayer()
{
	// Registering tests.
	m_tests["initializebufferfromuniformdistribution"] = bind(&TestLayer::TestInitializeBufferFromUniformDistribution, this);
	m_tests["initializebufferfromnormaldistribution"] = bind(&TestLayer::TestInitializeBufferFromNormalDistribution, this);
	m_tests["initializebuffertoconstant"] = bind(&TestLayer::TestInitializeBufferToConstant, this);
}

//******************************************************************************************************
// Tests
//******************************************************************************************************

bool TestLayer::TestInitializeBufferFromUniformDistribution()
{
	NeuralNet neuralNet(1);
	MockInputLayer mockInputLayer(1, 1000, 1, 128, neuralNet.GetCurandStatesBuffers()[0]);
	mockInputLayer.AllocateBuffers(false);
	Layer* mockInputLayerPt = static_cast<Layer*>(&mockInputLayer);

	const float rangeStart = -0.03f;
	const float rangeEnd = 0.07f;
	const uint activationDataBufferLength = (uint)(mockInputLayer.GetActivationBufferSize() / sizeof(float));
	mockInputLayerPt->InitializeBufferFromUniformDistribution(mockInputLayer.GetActivationDataBuffer(), activationDataBufferLength, rangeStart, rangeEnd, neuralNet.GetCurandStatesBuffers()[0]);
	mockInputLayerPt->SynchronizeCalculations();

	float* hostBuffer;
	CudaAssert(hipHostMalloc<float>(&hostBuffer, mockInputLayer.GetActivationBufferSize()));
	CudaAssert(hipMemcpy(hostBuffer, mockInputLayer.GetActivationDataBuffer(), mockInputLayer.GetActivationBufferSize(), hipMemcpyDeviceToHost));

	float minValue = hostBuffer[0];
	float maxValue = hostBuffer[0];
	bool foundDiffThanZero = false;
	for (size_t i = 1; i < activationDataBufferLength; ++i)
	{
		minValue = min(minValue, hostBuffer[i]);
		maxValue = max(maxValue, hostBuffer[i]);

		foundDiffThanZero = foundDiffThanZero || hostBuffer[i] != 0.f;
	}

	return minValue > rangeStart && maxValue <= rangeEnd && foundDiffThanZero;
}

bool TestLayer::TestInitializeBufferFromNormalDistribution()
{
	NeuralNet neuralNet(1);
	MockInputLayer mockInputLayer(1, 1000, 1, 128, neuralNet.GetCurandStatesBuffers()[0]);
	mockInputLayer.AllocateBuffers(false);
	Layer* mockInputLayerPt = static_cast<Layer*>(&mockInputLayer);

	const float mean = 0.05f;
	const float stDev = 0.02f;
	const uint activationDataBufferLength = (uint)(mockInputLayer.GetActivationBufferSize() / sizeof(float));
	mockInputLayerPt->InitializeBufferFromNormalDistribution(mockInputLayer.GetActivationDataBuffer(), activationDataBufferLength, mean, stDev, neuralNet.GetCurandStatesBuffers()[0]);
	mockInputLayerPt->SynchronizeCalculations();

	float* hostBuffer;
	CudaAssert(hipHostMalloc<float>(&hostBuffer, mockInputLayer.GetActivationBufferSize()));
	CudaAssert(hipMemcpy(hostBuffer, mockInputLayer.GetActivationDataBuffer(), mockInputLayer.GetActivationBufferSize(), hipMemcpyDeviceToHost));

	float minValue = hostBuffer[0];
	float maxValue = hostBuffer[0];
	float avgValue = 0.f;
	float cumAvgValue = hostBuffer[0];
	bool foundDiffThanZero = false;
	for (size_t i = 1; i < activationDataBufferLength; ++i)
	{
		minValue = min(minValue, hostBuffer[i]);
		maxValue = max(maxValue, hostBuffer[i]);
		cumAvgValue += hostBuffer[i];

		if (i % 1000 == 0 || i == (size_t)activationDataBufferLength - 1)
		{
			avgValue += cumAvgValue / activationDataBufferLength;
			cumAvgValue = 0.f;
		}

		foundDiffThanZero = foundDiffThanZero || hostBuffer[i] != 0.f;
	}

	return minValue < (mean - stDev) && maxValue > (mean + stDev) && abs(mean - avgValue) < 0.001f && foundDiffThanZero;

	return true;
}

bool TestLayer::TestInitializeBufferToConstant()
{
	NeuralNet neuralNet(1);
	MockInputLayer mockInputLayer(1, 1000, 1, 128, neuralNet.GetCurandStatesBuffers()[0]);
	mockInputLayer.AllocateBuffers(false);
	Layer* mockInputLayerPt = static_cast<Layer*>(&mockInputLayer);

	float initialValue = -0.7f;
	const uint activationDataBufferLength = (uint)(mockInputLayer.GetActivationBufferSize() / sizeof(float));
	mockInputLayerPt->InitializeBufferToConstant(mockInputLayer.GetActivationDataBuffer(), activationDataBufferLength, initialValue);
	mockInputLayerPt->SynchronizeCalculations();

	float* hostBuffer;
	CudaAssert(hipHostMalloc<float>(&hostBuffer, mockInputLayer.GetActivationBufferSize()));
	CudaAssert(hipMemcpy(hostBuffer, mockInputLayer.GetActivationDataBuffer(), mockInputLayer.GetActivationBufferSize(), hipMemcpyDeviceToHost));

	for (size_t i = 0; i < activationDataBufferLength; ++i)
	{
		if (hostBuffer[i] != initialValue)
		{
			return false;
		}
	}

	return true;
}