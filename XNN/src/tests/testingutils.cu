#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------------------------------
// Copyrighted by Marko Rakita.
// Author: Marko Rakita
// File contains: Utility functions for testing.
// Created: 12/12/2015.
// ----------------------------------------------------------------------------------------------------

#include "include/testingutils.cuh"

#include <algorithm>

void CompareBuffers(const float* regularBuffer, const float* mockBuffer, size_t buffersLength, float maxDiff, float maxDiffPercentage, float maxDiffPercentageThreshold,
	bool& correctResult, size_t& numDifferences, float& firstDifference, float& firstDifferentMock, float& firstDifferentReg, bool& foundDifferentFromZeroMock,
	bool& foundDifferentFromZeroReg)
{
	correctResult = true;
	numDifferences = 0;
	firstDifference = 0.f;
	firstDifferentMock = 0.f;
	firstDifferentReg = 0.f;
	foundDifferentFromZeroMock = false;
	foundDifferentFromZeroReg = false;

	for (size_t i = 0; i < buffersLength; ++i)
	{
		float diff = (float)fabs((double)mockBuffer[i] - regularBuffer[i]);
		if (diff > maxDiff || (diff > maxDiffPercentageThreshold && diff > maxDiffPercentage * max(abs(mockBuffer[i]), abs(regularBuffer[i]))))
		{
			++numDifferences;
			if (correctResult)
			{
				correctResult = false;
				firstDifference = mockBuffer[i] - regularBuffer[i];
				firstDifferentMock = mockBuffer[i];
				firstDifferentReg = regularBuffer[i];
			}
		}
		if (mockBuffer[i] != 0.0f && mockBuffer[i] != FLT_MIN)
		{
			foundDifferentFromZeroMock = true;
		}
		if (regularBuffer[i] != 0.0f && regularBuffer[i] != FLT_MIN)
		{
			foundDifferentFromZeroReg = true;
		}
	}
}