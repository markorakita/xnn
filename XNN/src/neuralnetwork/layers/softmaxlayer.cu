#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------------------------------
// Copyrighted by Marko Rakita.
// Author: Marko Rakita
// File contains: Neural network softmax layer.
// Created: 02/20/2016.
// ----------------------------------------------------------------------------------------------------

#include "include/softmaxlayer.cuh"

#include "include/outputlayer.cuh"
#include "../../utils/include/asserts.cuh"
#include "../../utils/include/config.cuh"
#include "../../utils/include/cudaasserts.cuh"
#include "../../utils/include/cudahelper.cuh"
#include "../../utils/include/utils.cuh"

SoftMaxLayer::SoftMaxLayer(ParallelismMode parallelismMode, hipStream_t deviceCalculationStream, hipStream_t deviceMemoryStream, uint inputDataSize,
	uint inputDataCount, bool holdsInputData)
{
	m_layerType = LayerType::SoftMax;
	m_parallelismMode = parallelismMode;
	m_deviceCalculationStream = deviceCalculationStream;
	m_deviceMemoryStream = deviceMemoryStream;
	m_indexInTier = 0;
	m_tierSize = 1;

	m_inputNumChannels = m_activationNumChannels = 1;
	m_inputDataWidth = m_activationDataWidth = inputDataSize;
	m_inputDataHeight = m_activationDataHeight = 1;
	m_inputDataSize = m_activationDataSize = inputDataSize;
	m_inputDataCount = inputDataCount;
	m_holdsInputData = holdsInputData;

	m_inputBufferSize = (size_t)m_inputDataSize * m_inputDataCount * sizeof(float);
	m_activationBufferSize = m_inputBufferSize;

	m_holdsActivationGradients = false;

	m_NLLsBuffer = NULL;
	m_inputActivationsMaxBuffer = NULL;
	m_exponentialsSumBuffer = NULL;
}

void SoftMaxLayer::AllocateBuffers(bool allocateTrainBuffers)
{
	CudaAssert(hipSetDevice(0));

	// Allocating input data buffer.
	if (m_holdsInputData)
	{
		CudaAssert(hipMalloc<float>(&m_inputDataBuffer, m_inputBufferSize));
		m_memoryConsumptionSize += m_inputBufferSize;
	}

	// Allocating input activations maximums buffer.
	size_t inputActivationsMaxBufferSize = m_inputDataCount * sizeof(float);
	CudaAssert(hipMalloc<float>(&m_inputActivationsMaxBuffer, inputActivationsMaxBufferSize));
	m_memoryConsumptionSize += inputActivationsMaxBufferSize;

	// Allocating sum of exponentials buffer.
	size_t exponentialsSumBufferSize = m_inputDataCount * sizeof(float);
	CudaAssert(hipMalloc<float>(&m_exponentialsSumBuffer, exponentialsSumBufferSize));
	m_memoryConsumptionSize += exponentialsSumBufferSize;

	// Allocating activation data buffers.
	CudaAssert(hipMalloc<float>(&m_activationDataBuffer, m_activationBufferSize));
	m_memoryConsumptionSize += m_activationBufferSize;

	// Allocating buffers necessary for training.
	if (allocateTrainBuffers)
	{
		// Allocating input gradients buffer.
		CudaAssert(hipMalloc<float>(&m_inputGradientsBuffer, m_inputBufferSize));
		m_memoryConsumptionSize += m_inputBufferSize;
	}
}

SoftMaxLayer::~SoftMaxLayer()
{
	if (m_inputActivationsMaxBuffer != NULL)
	{
		CudaAssert(hipFree(m_inputActivationsMaxBuffer));
	}
	if (m_exponentialsSumBuffer != NULL)
	{
		CudaAssert(hipFree(m_exponentialsSumBuffer));
	}
	if (m_NLLsBuffer != NULL)
	{
		CudaAssert(hipFree(m_NLLsBuffer));
	}
}

/*
	Finds maximum values of input activations for each input sample.
*/
__global__ void FindMaximums(float* inputActivations, const uint numInputSamples, const uint numInputActivations, float* inputActivationsMaximums)
{
	const uint c_sampleIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (c_sampleIndex < numInputSamples)
	{
		float activationMaximum = inputActivations[c_sampleIndex];
		for (uint activationIndex = 1; activationIndex < numInputActivations; ++activationIndex)
		{
			activationMaximum = max(activationMaximum, inputActivations[activationIndex * numInputSamples + c_sampleIndex]);
		}
		inputActivationsMaximums[c_sampleIndex] = activationMaximum;
	}
}

/*
	Subtracts maximum values of input activations from all input activations for each input sample.
*/
template <uint c_blockWidth>
__global__ void SubtractMaximums(float* inputActivations, const uint numInputSamples, const uint numInputActivations, float* inputActivationsMaximums,
	float* outputActivations)
{
	__shared__ float maximums[c_blockWidth];

	for (uint y = blockIdx.y * blockDim.y + threadIdx.y; y < numInputActivations; y += gridDim.y * blockDim.y)
	{
		__syncthreads();
		if (threadIdx.y == 0)
		{
			maximums[threadIdx.x] = inputActivationsMaximums[blockIdx.x * blockDim.x + threadIdx.x];
		}
		__syncthreads();

		const uint c_offset = y * numInputSamples;
		for (uint x = blockIdx.x * blockDim.x + threadIdx.x; x < numInputSamples; x += gridDim.x * blockDim.x)
		{
			outputActivations[c_offset + x] = inputActivations[c_offset + x] - maximums[threadIdx.x];
		}
	}
}

void SoftMaxLayer::StabilizeInputs()
{
	// Finding maximums of input activations.
	const uint c_numThreadsPerBlock = min((uint)Config::MAX_NUM_THREADS, RoundUp(m_inputDataCount, Config::WARP_SIZE));
	const uint c_numBlocks = DivideUp(m_inputDataCount, c_numThreadsPerBlock);
	LAUNCH_KERNEL_ASYNC(FindMaximums, dim3(c_numBlocks), dim3(c_numThreadsPerBlock), m_deviceCalculationStream)(m_inputDataBuffer, m_inputDataCount,
		m_activationDataSize, m_inputActivationsMaxBuffer);
	CudaAssert(hipGetLastError());

	// Substracting maximums of input activations from all the input activations.
	const uint c_blockWidth = 64;
	const uint c_blockHeight = (uint)Config::MAX_NUM_THREADS / c_blockWidth;
	dim3 blockDimensions(c_blockWidth, c_blockHeight);
	const uint c_maxGridBlocks = 128;
	const uint c_gridWidth = min(c_maxGridBlocks, DivideUp(m_inputDataCount, c_blockWidth));
	const uint c_gridHeight = min(c_maxGridBlocks / c_gridWidth, DivideUp(m_activationDataSize, c_blockHeight));
	dim3 gridDimensions(c_gridWidth, c_gridHeight);
	LAUNCH_KERNEL_ASYNC((SubtractMaximums<c_blockWidth>), gridDimensions, blockDimensions, m_deviceCalculationStream)(m_inputDataBuffer, m_inputDataCount,
		m_activationDataSize, m_inputActivationsMaxBuffer, m_activationDataBuffer);
	CudaAssert(hipGetLastError());
}

/*
	Computes the exponentials of activations.
*/
__global__ void ComputeExponentials(float* activations, const uint activationsLength)
{
	for (uint activationIndex = blockIdx.x * blockDim.x + threadIdx.x; activationIndex < activationsLength; activationIndex += gridDim.x * blockDim.x)
	{
		activations[activationIndex] = __expf(activations[activationIndex]);
	}
}

/*
	Computes sum of the exponentials of activations.
*/
__global__ void ComputeSumOfExponentials(float* activations, const uint numInputSamples, const uint numActivations, float* exponentialsSumBuffer)
{
	const uint c_sampleIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (c_sampleIndex < numInputSamples)
	{
		float exponentialsSum = 0.f;
		for (uint activationIndex = 0; activationIndex < numActivations; ++activationIndex)
		{
			exponentialsSum += activations[activationIndex * numInputSamples + c_sampleIndex];
		}
		exponentialsSumBuffer[c_sampleIndex] = exponentialsSum;
	}
}

/*
	Divides activation exponentials with their sum to get soft maximums.
*/
template <uint c_blockWidth>
__global__ void DivideExponentialsWithSum(float* activationExponentials, const uint numInputSamples, const uint numActivations, float* exponentialsSumBuffer)
{
	__shared__ float exponentialsSums[c_blockWidth];

	for (uint y = blockIdx.y * blockDim.y + threadIdx.y; y < numActivations; y += gridDim.y * blockDim.y)
	{
		__syncthreads();
		if (threadIdx.y == 0)
		{
			exponentialsSums[threadIdx.x] = exponentialsSumBuffer[blockIdx.x * blockDim.x + threadIdx.x];
		}
		__syncthreads();

		const uint c_offset = y * numInputSamples;
		for (uint x = blockIdx.x * blockDim.x + threadIdx.x; x < numInputSamples; x += gridDim.x * blockDim.x)
		{
			activationExponentials[c_offset + x] = __fdividef(activationExponentials[c_offset + x], exponentialsSums[threadIdx.x]);
		}
	}
}

void SoftMaxLayer::CalculateSoftMaximums()
{
	// Computing the exponentials.
	const uint c_activationBufferLength = (uint)(m_activationBufferSize / sizeof(float));
	uint numBlocks = 128;
	uint numThreadsPerBlock = 128;
	dim3 blockDimensions(numThreadsPerBlock);
	dim3 gridDimensions(min(numBlocks, DivideUp(c_activationBufferLength, numThreadsPerBlock)));
	LAUNCH_KERNEL_ASYNC(ComputeExponentials, gridDimensions, blockDimensions, m_deviceCalculationStream)(m_activationDataBuffer, c_activationBufferLength);
	CudaAssert(hipGetLastError());

	// Computing sum of the exponentials.
	numThreadsPerBlock = min((uint)Config::MAX_NUM_THREADS, RoundUp(m_inputDataCount, Config::WARP_SIZE));
	numBlocks = DivideUp(m_inputDataCount, numThreadsPerBlock);
	LAUNCH_KERNEL_ASYNC(ComputeSumOfExponentials, dim3(numBlocks), dim3(numThreadsPerBlock), m_deviceCalculationStream)(m_activationDataBuffer, m_inputDataCount,
		m_activationDataSize, m_exponentialsSumBuffer);
	CudaAssert(hipGetLastError());

	// Dividing exponentials with their sum to get soft maximums.
	const uint c_blockWidth = 64;
	const uint c_blockHeight = (uint)Config::MAX_NUM_THREADS / c_blockWidth;
	blockDimensions = dim3(c_blockWidth, c_blockHeight);
	const uint c_maxGridBlocks = 128;
	const uint c_gridWidth = min(c_maxGridBlocks, DivideUp(m_inputDataCount, c_blockWidth));
	const uint c_gridHeight = min(c_maxGridBlocks / c_gridWidth, DivideUp(m_activationDataSize, c_blockHeight));
	gridDimensions = dim3(c_gridWidth, c_gridHeight);
	LAUNCH_KERNEL_ASYNC((DivideExponentialsWithSum<c_blockWidth>), gridDimensions, blockDimensions, m_deviceCalculationStream)(m_activationDataBuffer, m_inputDataCount,
		m_activationDataSize, m_exponentialsSumBuffer);
	CudaAssert(hipGetLastError());
}

/*
	Calculates negative log likelihoods using LogSumExp formula.
*/
__global__ void __CalculateNegativeLogLikelihoods(float* inputActivations, uint* dataLabels, const uint numInputSamples, float* inputActivationsMaximums,
	float* exponentialsSumBuffer, float* nllsBuffer)
{
	const uint dataIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (dataIndex < numInputSamples)
	{
		nllsBuffer[dataIndex] = inputActivationsMaximums[dataIndex] + __logf(exponentialsSumBuffer[dataIndex]) -
			inputActivations[dataLabels[dataIndex] * numInputSamples + dataIndex];
	}
}

void SoftMaxLayer::CalculateNegativeLogLikelihoods(uint* dataLabels)
{
	if (m_NLLsBuffer == NULL)
	{
		size_t nllsBufferSize = m_inputDataCount * sizeof(float);
		CudaAssert(hipMalloc<float>(&m_NLLsBuffer, nllsBufferSize));
		m_memoryConsumptionSize += nllsBufferSize;
	}

	const uint numThreadsPerBlock = 128;
	const uint numBlocks = DivideUp(m_inputDataCount, numThreadsPerBlock);
	LAUNCH_KERNEL_ASYNC(__CalculateNegativeLogLikelihoods, dim3(numBlocks), dim3(numThreadsPerBlock), m_deviceCalculationStream)(m_inputDataBuffer,
		dataLabels, m_inputDataCount, m_inputActivationsMaxBuffer, m_exponentialsSumBuffer, m_NLLsBuffer);
	CudaAssert(hipGetLastError());
}

void SoftMaxLayer::DoForwardProp(PropagationMode propagationMode)
{
	StabilizeInputs();
	CalculateSoftMaximums();

	if (m_nextLayers[0]->GetLayerType() == LayerType::Output)
	{
		OutputLayer* outputLayer = static_cast<OutputLayer*>(m_nextLayers[0]);
		if (outputLayer->GetLossFunctionType() == LossFunctionType::CrossEntropy)
		{
			CalculateNegativeLogLikelihoods(outputLayer->GetDataLabels());
		}
		else
		{
			ShipAssert(false, "Currently not supported!");
		}
	}
}

/*
	Calculates input gradients in case of cross entropy loss in output layer.
*/
__global__ void CalculateCrossEntropyInputGradients(float* activations, uint* dataLabels, const uint dataCount, const uint numActivations,
	float* inputGradients)
{
	const uint c_dataIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const uint c_activationIndex = blockIdx.y * blockDim.y + threadIdx.y;
	const uint c_activationsOffset = c_activationIndex * dataCount + c_dataIndex;

	if (c_dataIndex < dataCount && c_activationIndex < numActivations)
	{
		inputGradients[c_activationsOffset] = (dataLabels[c_dataIndex] == c_activationIndex ? 1.f : 0.f) - activations[c_activationsOffset];
	}
}

void SoftMaxLayer::CrossEntropyBackwardProp(uint* dataLabels)
{
	const uint c_blockWidth = 32;
	const uint c_blockHeight = 4;
	dim3 blockDimensions(c_blockWidth, c_blockHeight);
	const uint c_gridWidth = DivideUp(m_inputDataCount, c_blockWidth);
	const uint c_gridHeight = DivideUp(m_activationDataSize, c_blockHeight);
	dim3 gridDimensions(c_gridWidth, c_gridHeight);
	LAUNCH_KERNEL_ASYNC(CalculateCrossEntropyInputGradients, gridDimensions, blockDimensions, m_deviceCalculationStream)(m_activationDataBuffer,
		dataLabels, m_inputDataCount, m_activationDataSize, m_inputGradientsBuffer);
	CudaAssert(hipGetLastError());
}

void SoftMaxLayer::DoBackwardProp()
{
	if (m_nextLayers[0]->GetLayerType() == LayerType::Output)
	{
		OutputLayer* outputLayer = static_cast<OutputLayer*>(m_nextLayers[0]);
		if (outputLayer->GetLossFunctionType() == LossFunctionType::CrossEntropy)
		{
			CrossEntropyBackwardProp(outputLayer->GetDataLabels());
		}
		else
		{
			ShipAssert(false, "Currently not supported!");
		}
	}
	else
	{
		ShipAssert(false, "Currently not supported!");
	}
}